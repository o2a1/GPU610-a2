//Calculate prime numbers within a certain range provided by the user, or use default
//values of 0-1000

#include <iostream>
#include <cstdint>	//required for uint64_t
#include <sstream>	//convert runtime params into uint64 using istringstream
#include <ctime>
#include <chrono>
#include <cstdlib>
#include <hip/hip_runtime.h>
const int MAX_THREADS = 1024;
using namespace std::chrono;

/*double inline __declspec (naked) __fastcall sqrt(double n)
{
_asm fld qword ptr[esp + 4]
_asm fsqrt
_asm ret 8
}*/

void reportTime(const char* msg, steady_clock::duration span) {
	auto ms = duration_cast<milliseconds>(span);
	std::cout << msg << " - took - " <<
		ms.count() << " millisecs" << std::endl;
}

uint64_t genPrime(uint64_t a, uint64_t b) {
	//Keep track of results
	uint64_t count = 0;
	//Outer loop
	for (uint64_t i = a; i < b; i++)
		//Inner loop
		for (uint64_t j = 2; j*j <= i; j++) {
			if (i % j == 0)
				break;
			else if (j + 1 > sqrt(i)) {
				//Actual output
				//std::cout.precision(0);
				std::cout << std::fixed << i << "\n";
				count++;
			}
		}
	return count;	//Return total number of primes generated in the range specified
}

int main(int argc, char* argv[]) {
	std::cout << "***Team /dev/null GPU610 PRIME NUMBER GENERATOR v1.2***\n";

	//In case the user didn't provide arguments
	uint64_t start = 0; //orig 21474836470000
	uint64_t end = 1000;  //orig 214748364700000

	//Save runtime params into local variables, if provided
	if (argc == 2) {
		std::istringstream ss1(argv[1]);
		if (!(ss1 >> end))
			std::cout << "Bad input for end parameter\n";
	}

	if (argc == 3) {
		std::istringstream ss2(argv[1]);
		if (!(ss2 >> start))
			std::cout << "Bad input for start parameter\n";

		std::istringstream ss3(argv[2]);
		if (!(ss3 >> end))
			std::cout << "Bad input for end parameter\n";
	}
	else std::cout << "No range given (or bad input), using preset values\n";
	if (start >= end) {
		std::cerr << "***Invalid input, start must be less than end***\n";
	}
	std::cout << "Generating from range (" << start << "~" << end << ")\n";
	std::cout << "--------------------------------------------------------------------------------\n";
	//Keep track of time spent doing calculations
	steady_clock::time_point ts, te;
	ts = steady_clock::now();

	//Generate primes
	//Starting with the thread calculations
	uint64_t threadStart = 0; //The first portion, where each thread begins
	uint64_t threadEnd, taskLength;
	int threadAmount = MAX_THREADS; //Replace MAX_THREADS with amount of threads once we find the function
	taskLength = (end - start) / threadAmount; //Assigns the length of each portion of the task. This is how much of the total function runs in each thread.
	threadEnd = taskLength;
	//CUDA Allocation (please freaking work)
	double* h_a = new double[start];
	double* h_b = new double[end];
	double* d_a, d_b;
	hipMalloc((void**)&d_a, taskLength * sizeof(double));

	//End cuda allocation

	//Function call

	uint64_t count = genPrime(start, end);	//REPLACE WITH CUDA KERNEL CALL
	te = steady_clock::now();

	std::cout << "\n--------------------------------------------------------------------------------\n"
		<< "There are " << count << " prime numbers in the calculated range.\n";
	reportTime("Took: {0} seconds", te - ts);
	return 0;
}

/* Original code
int main()
{
for (int i = 2; i<100; i++)
for (int j = 2; j*j <= i; j++)
{
if (i % j == 0)
break;
else if (j + 1 > sqrt(i)) {
std::cout << i << " ";
}
}
return 0;
}
*/

//Changelog
/*
v1 - Generating from simple double loop
v1.0.1 - Command line parameter input
v1.1 - Nicer output format and error feedback
v1.2 - Full 64 bit integer compatibility

*/

/*TODO
	- Write kernel function to replace genPrime()
	-complete/correct CUDA memory and thread allocation
	-Write new function call for genPrime<<<>>> //The <<<>>> mean kernel function
	-... Yeah that's it.
	
*/